
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_complex.h>


#define PI 3.14159265358979323846
//Las muestras están entre -1 y 1
#define T_MAX 1
#define T_MIN -1
#define BLOCK_SIZE 256
#define NUM_BLOCKS 10

//Si no hay exponencial en cuda la creo yo (Informacion cogida de la misma pagina de CUDA: https://forums.developer.nvidia.com/t/additional-cucomplex-functions-cucnorm-cucsqrt-cucexp-and-some-complex-double-functions/36892)
__host__ __device__ static __inline__ hipDoubleComplex cuCexp(hipDoubleComplex x)
{
	double factor = exp(x.x);
	return make_hipDoubleComplex(factor * cos(x.y), factor * sin(x.y));
}

//Esta si que la he puesto yo como me gusta, pero tambine viene en la pagina como un CuMul normal para los complejos dobles

__host__ __device__ static __inline__ hipDoubleComplex cuCmulReal(hipDoubleComplex a, double r) {
    return make_hipDoubleComplex(a.x * r, a.y * r);
}


//Aqui mis funciones 
__global__ void DFT(hipDoubleComplex *Fourier, const double *muestras, const int TAM_VECTOR_MUESTRAS){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < TAM_VECTOR_MUESTRAS){
        Fourier[i] = make_hipDoubleComplex(0.0,0.0);
        hipDoubleComplex sum = make_hipDoubleComplex(0.0,0.0);
        for (int j=0;j<TAM_VECTOR_MUESTRAS;j++){
            double angle = -2.0*PI*i*j/TAM_VECTOR_MUESTRAS;
            hipDoubleComplex aux = make_hipDoubleComplex(muestras[j]*cos(angle),muestras[j]*sin(angle));
            sum = hipCadd(sum,aux);
        }
        Fourier[i] = sum;
    }    
}

__global__ void CFT(hipDoubleComplex *Fourier, const double *muestras, const int TAM_VECTOR_MUESTRAS, const double paso_temporal){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < TAM_VECTOR_MUESTRAS){
        Fourier[i] = make_hipDoubleComplex(0.0,0.0);
        hipDoubleComplex sum = make_hipDoubleComplex(0.0,0.0);
        double omega = 2.0*PI*i/(T_MAX-T_MIN);//La w de la formula que es el omega
        //Aqui ya entra en juego tanto el intervalo del tiempo como los valores que dan la funcion, es decir el tiempo esta entre -1 y 1
        //y los valores de la funcion están en mis muestras
        //Vamos desde el minimo hasta el maximo pero con nuestro paso temporal para tomar fourier lo más preciso posible
        for (double j = T_MIN;j<T_MAX; j= j+paso_temporal){
            //printf("Estoy en el segundo FOR");
            int indice = (int)((j - T_MIN)/paso_temporal);
            if (indice <= 0 ){
                // Si es menor o igual a 0, suponemos que coge el primer elemento
                hipDoubleComplex expo = cuCexp(make_hipDoubleComplex(0.0, -omega * j));  
                hipDoubleComplex temp = make_hipDoubleComplex(muestras[0], 0.0);  
                hipDoubleComplex prod = hipCmul(temp, expo);  
                sum = hipCadd(sum, cuCmulReal(prod, paso_temporal));  
            }
            else if (indice >= TAM_VECTOR_MUESTRAS - 1) { 
                // Si es mayor o igual al número de elementos, cogemos el último
                hipDoubleComplex expo = cuCexp(make_hipDoubleComplex(0.0, -omega * j));  
                hipDoubleComplex temp = make_hipDoubleComplex(muestras[TAM_VECTOR_MUESTRAS - 1], 0.0);  
                hipDoubleComplex prod = hipCmul(temp, expo);  
                sum = hipCadd(sum, cuCmulReal(prod, paso_temporal));  
            } else {
                // Si no es válido y cogemos el valor calculado
                hipDoubleComplex expo = cuCexp(make_hipDoubleComplex(0.0, -omega * j));  // Usamos la función cuCexp para la exponencial
                hipDoubleComplex temp = make_hipDoubleComplex(muestras[indice], 0.0);  // Tomamos la muestra correspondiente
                hipDoubleComplex prod = hipCmul(temp, expo);  // Multiplicamos la muestra por la exponencial
                sum = hipCadd(sum, cuCmulReal(prod, paso_temporal));  // Acumulamos el resultado, aplicando el paso temporal
            }
        }
        Fourier[i] = sum;
    }    
}

__global__ void CFT_Simpson(hipDoubleComplex *Fourier, const double *muestras, const int TAM_VECTOR_MUESTRAS, const double paso_temporal){

        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < TAM_VECTOR_MUESTRAS){
            Fourier[i] = make_hipDoubleComplex(0.0,0.0);
            hipDoubleComplex sum = make_hipDoubleComplex(0.0,0.0);
            double omega = 2.0*PI*i/(T_MAX-T_MIN);//La w de la formula que es el omega

            for (double j = T_MIN;j<T_MAX - paso_temporal; j += paso_temporal){
                int indice1 = (int)((j - T_MIN)/paso_temporal);
                int indice2 = indice1 + 1;

                if (indice2 >= TAM_VECTOR_MUESTRAS) indice2 = TAM_VECTOR_MUESTRAS - 1;
                
                double x_medio = j + paso_temporal / 2.0;
                int indice_medio = (int)((x_medio - T_MIN) / paso_temporal);

                if (indice_medio >= TAM_VECTOR_MUESTRAS) indice_medio = TAM_VECTOR_MUESTRAS - 1;

                double simpson = (muestras[indice1] + 4.0*muestras[indice_medio] + muestras[indice2])/6.0;

                //Aqui la parte nueva además de distribuirlo para cada 
                hipDoubleComplex prod = cuCexp(make_hipDoubleComplex(0.0, -omega * j));
                hipDoubleComplex temp = make_hipDoubleComplex(simpson, 0.0);
                hipDoubleComplex prod2 = hipCmul(temp, prod);
                sum = hipCadd(sum,cuCmulReal(prod2,paso_temporal));


            }

            Fourier[i] = sum;

        }


}

__global__ void CFT_Trapecio(hipDoubleComplex *Fourier, const double *muestras, const int TAM_VECTOR_MUESTRAS, const double paso_temporal){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < TAM_VECTOR_MUESTRAS){
        Fourier[i] = make_hipDoubleComplex(0.0,0.0);
        hipDoubleComplex sum = make_hipDoubleComplex(0.0,0.0);
        double omega = 2.0*PI*i/(T_MAX-T_MIN);//La w de la formula que es el omega

        for(double j = T_MIN; j < T_MAX - paso_temporal; j += paso_temporal){
            int indice1 = (int)((j - T_MIN) / paso_temporal);
            int indice2 = indice1 + 1;

            if (indice2 >= TAM_VECTOR_MUESTRAS) indice2 = TAM_VECTOR_MUESTRAS - 1;

            double promedio = (muestras[indice1] + muestras[indice2])/2.0;

            //Aqui la parte nueva además de distribuirlo para cada
            hipDoubleComplex prod = cuCexp(make_hipDoubleComplex(0.0, -omega * j));
            hipDoubleComplex temp = make_hipDoubleComplex(promedio, 0.0);
            hipDoubleComplex prod2 = hipCmul(temp, prod);
            sum = hipCadd(sum,cuCmulReal(prod2,paso_temporal));

        }

        Fourier[i] = sum;

    }


}


//Apartador de archivos de entrada y saluda
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
const char * MILLON = "txt/MuestraGenerada.txt"; //una muestra de un millo de elementos 🫨🫨
const char * TREINTAMIL = "txt/MuestraGenerada30000.txt"; //una muestra de un 30.000 elementos 🫨🫨
const char * CINCUENTAMIL = "txt/MuestraGenerada50000.txt";
const char * CIENMIL = "txt/MuestraGenerada100000.txt";
const char * CIENTOCINCUENTAMIL = "txt/MuestraGenerada150000.txt";
const char * DOSCIENTOSCINCUENTAMIL = "txt/MuestraGenerada250000.txt";
const char * QUINIENTOSMIL = "txt/MuestraGenerada500000.txt";
const char * DISTINTAS20 = "txt/muestras.txt"; //20.000 muestras de elementos aleatorios 😎😎
const char * FUNCIONA = "txt/funciona.txt"; //muestras de funcionamiento 😎😎
const char * FUNCIONA2 = "txt/funciona2.txt"; //muestras de funcionamiento 😎😎
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
const char * SALIDA = "Resultados/SecuencialCUDADFT.txt"; //salida secuencial (cualquier caso) 😇😇
const char * SALIDA2 = "txt/SecuencialDFT2.txt"; //salida secuencial para el caso de 20.000 muestras 😇😇
const char * SALIDA_CONTINUO = "Resultados/SecuencialCUDACFTNormal.txt"; //salida continuo 🤯🤯
const char * SALIDA_CONTINUO2 = "Resultados/SecuencialCUDACFTSimpson.txt"; //salida continuo 🤯🤯
const char * SALIDA_CONTINUO3 = "Resultados/SecuencialCUDACFTTrapecio.txt"; //salida continuo 🤯🤯
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main (){

FILE * entrada = fopen (QUINIENTOSMIL,"r");
    FILE * salida = fopen(SALIDA,"w");
    FILE * salida_cont = fopen(SALIDA_CONTINUO,"w");
    FILE * salida_cont2 = fopen(SALIDA_CONTINUO2,"w");
    FILE * salida_cont3 = fopen(SALIDA_CONTINUO3,"w");

    if (!entrada){
        printf("Error: No se pudo abrir el archivo de entrada\n");
        exit(1);
    }
    if (!salida){
        printf("Error: No se pudo abrir el archivo de salida\n");
        exit(1);
    }

    if (!salida_cont){
        printf("Error: No se pudo abrir el archivo de salida continuo\n");
        exit(1);
    }
    if (!salida_cont2){
        printf("Error: No se pudo abrir el archivo de salida continuo\n");
        exit(1);
    }
    if (!salida_cont3){
        printf("Error: No se pudo abrir el archivo de salida continuo\n");
        exit(1);
    }
    
    int Tam_Vector_muestras;
    double *muestras;
    hipDoubleComplex *Fourier;
    hipDoubleComplex *Fourier_cont;
    hipDoubleComplex *Fourier_cont2;
    hipDoubleComplex *Fourier_cont3;
    while (fscanf(entrada,"%d",&Tam_Vector_muestras) == 1){
        
        hipMallocManaged(&muestras,Tam_Vector_muestras*sizeof(double));//double *muestras = malloc(Tam_Vector_muestras*sizeof(double));
        hipMallocManaged(&Fourier ,Tam_Vector_muestras*sizeof(hipDoubleComplex) );//double complex *Fourier = malloc(Tam_Vector_muestras*sizeof(double complex));
        hipMallocManaged(&Fourier_cont ,Tam_Vector_muestras*sizeof(hipDoubleComplex) );//double complex *Fourier_cont = malloc(Tam_Vector_muestras*sizeof(double complex));
        hipMallocManaged(&Fourier_cont2 ,Tam_Vector_muestras*sizeof(hipDoubleComplex) );//double complex *Fourier_cont2 = malloc(Tam_Vector_muestras*sizeof(double complex));
        hipMallocManaged(&Fourier_cont3 ,Tam_Vector_muestras*sizeof(hipDoubleComplex) );//double complex *Fourier_cont3 = malloc(Tam_Vector_muestras*sizeof(double complex));
        
        if (!muestras){
            printf("Error: No se pudo asignar memoria muestras\n");
            exit(1);
        }
        if (!Fourier){
            printf("Error: No se pudo asignar memoria Fourier\n");
            exit(1);
        }
        if (!Fourier_cont){
            printf("Error: No se pudo asignar memoria Fourier\n");
            exit(1);
        }
        if (!Fourier_cont2){
            printf("Error: No se pudo asignar memoria Fourier\n");
            exit(1);
        }
        if (!Fourier_cont3){
            printf("Error: No se pudo asignar memoria Fourier\n");
            exit(1);
        }

        for (int i=0;i<Tam_Vector_muestras;i++){
            if (fscanf(entrada,"%lf",&muestras[i]) != 1){
                printf("Error: archivo de entrada de la muestras %d\n",i);
                exit(1);
            }
        }

        //////////////////////////////////////////////////////////////////////////////////////////////////////////
        //Parte de DFT
        //////////////////////////////////////////////////////////////////////////////////////////////////////////

        hipEvent_t start, stop;
        float milliseconds = 0;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0);
        DFT<<<NUM_BLOCKS,BLOCK_SIZE>>>(Fourier,muestras,Tam_Vector_muestras);
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        
        hipEventElapsedTime(&milliseconds, start, stop);
        
        fprintf(salida,"%d %lf\n",Tam_Vector_muestras,milliseconds);
        //Para mostrar el vector, esta dentro del archivo 😴😴
        /*for (int i=0;i<Tam_Vector_muestras;i++){
            fprintf(salida, "%lf %lf\n", cuCreal(Fourier[i]), cuCimag(Fourier[i]));
        }*/

        hipEventDestroy(start);
        hipEventDestroy(stop);

        //////////////////////////////////////////////////////////////////////////////////////////////////////////
        //Parte de CFT
        //////////////////////////////////////////////////////////////////////////////////////////////////////////
        //printf("valor tammuestras %d\n",Tam_Vector_muestras);
        //printf("valor tmax - tmin %d\n",T_MAX-T_MIN);
        //printf("valor paso temporal %f\n", (double)(T_MAX-T_MIN) / Tam_Vector_muestras);
        
        
        double paso_temporal = (double)(T_MAX-T_MIN) / Tam_Vector_muestras;
        hipEvent_t start1, stop1;
        milliseconds = 0;
        hipEventCreate(&start1);
        hipEventCreate(&stop1);


        hipEventRecord(start1, 0);
        CFT<<<NUM_BLOCKS,BLOCK_SIZE>>>(Fourier_cont,muestras,Tam_Vector_muestras,paso_temporal);
        hipEventRecord(stop1, 0);
        hipEventSynchronize(stop1);

        hipEventElapsedTime(&milliseconds, start1, stop1);
        
        //printf("Valor paso temporal: %f\n",paso_temporal);
        //clock_t inicio_cont = clock();
        //CFT(Fourier_cont,muestras,Tam_Vector_muestras,paso_temporal);
        //clock_t fin_cont = clock();
        //double tiempo_cont = (double)(fin_cont-inicio_cont)/CLOCKS_PER_SEC;
        fprintf(salida_cont,"%d %lf\n",Tam_Vector_muestras,milliseconds);
        /*for (int i=0;i<Tam_Vector_muestras;i++){
            fprintf(salida_cont, "%lf %lf\n", cuCreal(Fourier_cont[i]), cuCimag(Fourier_cont[i]));
        }*/

        //printf("Valor paso temporal: %f\n",paso_temporal);
        /*clock_t inicio_cont2 = clock();
        CFT_Simpson(Fourier_cont2,muestras,Tam_Vector_muestras,paso_temporal);
        clock_t fin_cont2 = clock();
        double tiempo_cont2 = (double)(fin_cont2-inicio_cont2)/CLOCKS_PER_SEC;
        fprintf(salida_cont2,"%d %lf\n",Tam_Vector_muestras,tiempo_cont2);
        /*for (int i=0;i<Tam_Vector_muestras;i++){
            fprintf(salida_cont2,"%lf %lf\n",creal(Fourier_cont2[i]),cimag(Fourier_cont2[i]));
        }*/

        hipEvent_t start2, stop2;
        float milliseconds2 = 0;
        hipEventCreate(&start2);
        hipEventCreate(&stop2);

        hipEventRecord(start2, 0);
        CFT_Simpson<<<NUM_BLOCKS,BLOCK_SIZE>>>(Fourier_cont2,muestras,Tam_Vector_muestras,paso_temporal);
        hipEventRecord(stop2, 0);
        hipEventSynchronize(stop2);

        hipEventElapsedTime(&milliseconds2, start2, stop2);
        
        fprintf(salida_cont2,"%d %lf\n",Tam_Vector_muestras,milliseconds2);
        /*for (int i=0;i<Tam_Vector_muestras;i++){
            fprintf(salida_cont2, "%lf %lf\n", cuCreal(Fourier_cont2[i]), cuCimag(Fourier_cont2[i]));
        }*/



        
        hipEvent_t start3, stop3;
        float milliseconds3 = 0;
        hipEventCreate(&start3);
        hipEventCreate(&stop3);

        hipEventRecord(start3, 0);
        CFT_Trapecio<<<NUM_BLOCKS,BLOCK_SIZE>>>(Fourier_cont3,muestras,Tam_Vector_muestras,paso_temporal);
        hipEventRecord(stop3, 0);
        hipEventSynchronize(stop3);

        hipEventElapsedTime(&milliseconds3, start3, stop3);
        
        fprintf(salida_cont3,"%d %lf\n",Tam_Vector_muestras,milliseconds3);
        /*for (int i=0;i<Tam_Vector_muestras;i++){
            fprintf(salida_cont3, "%lf %lf\n", cuCreal(Fourier_cont3[i]), cuCimag(Fourier_cont3[i]));
        }*/

        


        //printf("Valor paso temporal: %f\n",paso_temporal);
        /*clock_t inicio_cont3 = clock();
        CFT_Trapecio(Fourier_cont3,muestras,Tam_Vector_muestras,paso_temporal);
        clock_t fin_cont3 = clock();
        double tiempo_cont3 = (double)(fin_cont3-inicio_cont3)/CLOCKS_PER_SEC;
        fprintf(salida_cont3,"%d %lf\n",Tam_Vector_muestras,tiempo_cont3);
        /*for (int i=0;i<Tam_Vector_muestras;i++){
            fprintf(salida_cont3,"%lf %lf\n",creal(Fourier_cont3[i]),cimag(Fourier_cont3[i]));
        }*/
        //////////////////////////////////////////////////////////////////////////////////////////////////////////

        hipFree(muestras);
        hipFree(Fourier);
        hipFree(Fourier_cont);
        hipFree(Fourier_cont2);
        hipFree(Fourier_cont3);

    }

    fclose(entrada);
    fclose(salida);
    fclose(salida_cont);
    fclose(salida_cont2);
    fclose(salida_cont3);

    printf("Fin del programa, resultados guardados con formato NumeroMuestras Tiempo(Fourier Discreto) en salidaDFT\n");
    printf("Fin del programa, resultados guardados con formato NumeroMuestras Tiempo(Fourier continuo) en salidaCFT\n");
    printf("Fin del programa, resultados guardados con formato NumeroMuestras Tiempo(Fourier continuo Simpson) en salidaCFT\n");
    printf("Fin del programa, resultados guardados con formato NumeroMuestras Tiempo(Fourier continuo Trapezio) en salidaCFT\n");

    return 0;
    






}
